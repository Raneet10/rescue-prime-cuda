#include <iostream>
#include <hip/hip_runtime_api.h>
#include "rescue-prime.cuh"

int main() {
    int64_t *cmds = nullptr, *cround_constants = nullptr
    checkCudaErrors(hipMalloc(&cmds, sizeof(rescue_prime::mds)));
    checkCudaErrors(hipMalloc(&cround_constants, sizeof(rescue_prime::round_constants)));
    checkCudaErrors(hipMemcpy(cmds, mds, sizeof(rescue_prime::mds), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(cround_constants, rescue_prime::round_constants, sizeof(rescue_prime::round_constants), hipMemcpyHostToDevice));
    dim3 gridDim(32*m*sizeof(__uint128_t), 32*m*sizeof(__uint128_t));
    dim3 blockDim(32, 32);
    rescue_prime::rescuePrime<__uint128_t><<<gridDim, blockDim>>>(input_sequence, N, m, cmds, cround_constants, alpha, alphainv, rate);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipFree(cmds));
    checkCudaErrors(hipFree(cround_constants));
    return 0;
}